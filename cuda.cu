#include "hip/hip_runtime.h"
/*�˳���Ĭ����������Ϊ��1����ÿ��body������Ϊ��1��*/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float4 *pos, *vel; } BodySystem;

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n, int div)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < div)
	{//���㡰i��������
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

		for (int tile = 0; tile < gridDim.x; tile++)
		{
			__shared__ float3 spos[BLOCK_SIZE]; //�����ڴ�
			float4 tpos = p[tile * blockDim.x + threadIdx.x];
			spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
			__syncthreads(); //�����߳�ͬ��

			for (int j = 0; j < BLOCK_SIZE; j++)
			{
				float dx = spos[j].x - p[i].x;
				float dy = spos[j].y - p[i].y;
				float dz = spos[j].z - p[i].z;
				float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
				float invDist = rsqrtf(distSqr);
				float invDist3 = invDist * invDist * invDist;

				Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3; // ����x,y,z���������
			}
			__syncthreads(); //�����߳�ͬ��
		}

		v[i].x += dt * Fx; v[i].y += dt * Fy; v[i].z += dt * Fz; // ����x,y,z���������
	}
}

extern "C" void cumain(int nBodies, float *h_buf, float dt, int div);
extern "C" void cumain(int nBodies, float *h_buf, float dt, int div)
{
	int bytes0 = 2 * nBodies * sizeof(float4);
	int bytes1 = 2 * div * sizeof(float4);

	float *d_buf;
	hipMalloc(&d_buf, bytes0);
	BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

	int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;

	hipMemcpy(d_buf, h_buf, bytes0, hipMemcpyHostToDevice);
	bodyForce << <nBlocks, BLOCK_SIZE >> > (d_p.pos, d_p.vel, dt, nBodies, div);
	hipMemcpy(h_buf, d_buf, bytes1, hipMemcpyDeviceToHost);

	hipFree(d_buf);
}
